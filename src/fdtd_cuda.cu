#include "hip/hip_runtime.h"
#include "fdtd_cuda.hpp"
#include <math.h>
#include <algorithm>
#undef NDEBUG
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/* Cuda TODO:
  o  check all cuda routines for error
  o  3D array allocation using cudaMallod3D() ?
  o 

*/

__constant__ fdtd::CudaData cd;

fdtd::FDTD::FDTD(int Nx, int Ny, int Nz)
{
    _Nx = Nx;
    _Ny = Ny;
    _Nz = Nz;

	// Allocate field arrays
	int N = _Nz * _Ny * _Nx;
	hipMallocManaged((void **)&_hcd.Ex, N*sizeof(double));
	hipMallocManaged((void **)&_hcd.Ey, N*sizeof(double));
	hipMallocManaged((void **)&_hcd.Ez, N*sizeof(double));
	hipMallocManaged((void **)&_hcd.Hx, N*sizeof(double));
	hipMallocManaged((void **)&_hcd.Hy, N*sizeof(double));
	hipMallocManaged((void **)&_hcd.Hz, N*sizeof(double));

	// Allocate material arrays
	hipMallocManaged((void **)&_hcd.eps_x, N*sizeof(complex128));
	hipMallocManaged((void **)&_hcd.eps_y, N*sizeof(complex128));
	hipMallocManaged((void **)&_hcd.eps_z, N*sizeof(complex128));
	hipMallocManaged((void **)&_hcd.mu_x, N*sizeof(complex128));
	hipMallocManaged((void **)&_hcd.mu_y, N*sizeof(complex128));
	hipMallocManaged((void **)&_hcd.mu_z, N*sizeof(complex128));

    // make sure all of our PML arrays start NULL
    _hcd.pml_Exy0 = NULL; _hcd.pml_Exy1 = NULL; _hcd.pml_Exz0 = NULL; _hcd.pml_Exz1 = NULL;
    _hcd.pml_Eyx0 = NULL; _hcd.pml_Eyx1 = NULL; _hcd.pml_Eyz0 = NULL; _hcd.pml_Eyz1 = NULL;
    _hcd.pml_Ezx0 = NULL; _hcd.pml_Ezx1 = NULL; _hcd.pml_Ezy0 = NULL; _hcd.pml_Ezy1 = NULL;
    _hcd.pml_Hxy0 = NULL; _hcd.pml_Hxy1 = NULL; _hcd.pml_Hxz0 = NULL; _hcd.pml_Hxz1 = NULL;
    _hcd.pml_Hyx0 = NULL; _hcd.pml_Hyx1 = NULL; _hcd.pml_Hyz0 = NULL; _hcd.pml_Hyz1 = NULL;
    _hcd.pml_Hzx0 = NULL; _hcd.pml_Hzx1 = NULL; _hcd.pml_Hzy0 = NULL; _hcd.pml_Hzy1 = NULL;

    _hcd.kappa_H_x = NULL; _hcd.kappa_H_y = NULL; _hcd.kappa_H_z = NULL;
    _hcd.kappa_E_x = NULL; _hcd.kappa_E_y = NULL; _hcd.kappa_E_z = NULL;

    _hcd.bHx = NULL; _hcd.bHy = NULL; _hcd.bHz = NULL;
    _hcd.bEx = NULL; _hcd.bEy = NULL; _hcd.bEz = NULL;

    _hcd.cHx = NULL; _hcd.cHy = NULL; _hcd.cHz = NULL;
    _hcd.cEx = NULL; _hcd.cEy = NULL; _hcd.cEz = NULL;

    _w_pml_x0 = 0; _w_pml_x1 = 0;
    _w_pml_y0 = 0; _w_pml_y1 = 0;
    _w_pml_z0 = 0; _w_pml_z1 = 0;
}

fdtd::FDTD::~FDTD()
{
	// Clean up Field arrays
	hipFree(_hcd.Ex); hipFree(_hcd.Ey); hipFree(_hcd.Ez);
	hipFree(_hcd.Hx); hipFree(_hcd.Hy); hipFree(_hcd.Hz);

	// Clean up Material arrays
	hipFree(_hcd.eps_x); hipFree(_hcd.eps_y); hipFree(_hcd.eps_z);
	hipFree(_hcd.mu_x); hipFree(_hcd.mu_y); hipFree(_hcd.mu_z);

    // Clean up PML arrays
    hipFree(_hcd.pml_Exy0); hipFree(_hcd.pml_Exy1); hipFree(_hcd.pml_Exz0); hipFree(_hcd.pml_Exz1);
    hipFree(_hcd.pml_Eyx0); hipFree(_hcd.pml_Eyx1); hipFree(_hcd.pml_Eyz0); hipFree(_hcd.pml_Eyz1);
    hipFree(_hcd.pml_Ezx0); hipFree(_hcd.pml_Ezx1); hipFree(_hcd.pml_Ezy0); hipFree(_hcd.pml_Ezy1);
    hipFree(_hcd.pml_Hxy0); hipFree(_hcd.pml_Hxy1); hipFree(_hcd.pml_Hxz0); hipFree(_hcd.pml_Hxz1);
    hipFree(_hcd.pml_Hyx0); hipFree(_hcd.pml_Hyx1); hipFree(_hcd.pml_Hyz0); hipFree(_hcd.pml_Hyz1);
    hipFree(_hcd.pml_Hzx0); hipFree(_hcd.pml_Hzx1); hipFree(_hcd.pml_Hzy0); hipFree(_hcd.pml_Hzy1);

    hipFree(_hcd.kappa_H_x);
    hipFree(_hcd.kappa_H_y);
    hipFree(_hcd.kappa_H_z);

    hipFree(_hcd.kappa_E_x);
    hipFree(_hcd.kappa_E_y);
    hipFree(_hcd.kappa_E_z);

    hipFree(_hcd.bHx);
    hipFree(_hcd.bHy);
    hipFree(_hcd.bHz);

    hipFree(_hcd.bEx);
    hipFree(_hcd.bEy);
    hipFree(_hcd.bEz);

    hipFree(_hcd.cHx);
    hipFree(_hcd.cHy);
    hipFree(_hcd.cHz);

    hipFree(_hcd.cEx);
    hipFree(_hcd.cEy);
    hipFree(_hcd.cEz);
}

void fdtd::FDTD::set_physical_dims(double X, double Y, double Z,
								   double dx, double dy, double dz)
{
    _X = X; _Y = Y; _Z = Z;
    _dx = dx; _dy = dy; _dz = dz;
}

void fdtd::FDTD::set_wavelength(double wavelength)
{
    _wavelength = wavelength;
    _R = _wavelength/(2*M_PI);
}


void fdtd::FDTD::set_dt(double dt)
{
    _hcd.dt = dt;
}

__device__
double cuda_src_func_t(double t, double phase)
{
    if(t <= cd.src_T)
        return sin(t + phase)*((1+cd.src_min) * exp(-(t-cd.src_T)*(t-cd.src_T) / cd.src_k) - cd.src_min);
    else
        return sin(t + phase);
}

__global__
void update_H_fields(double t)
{
	double b, C, kappa,	dt_by_mux, dt_by_muy, dt_by_muz;

    int ind_ijk, ind_ip1jk, ind_ijp1k, ind_ijkp1,
        ind_pml, 
        ind_pml_param;

    double dExdy, dExdz, dEydx, dEydz, dEzdx, dEzdy;

	int i = blockIdx.z * blockDim.z + threadIdx.z;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int Nx = cd.Nx;
	int Ny = cd.Ny;
	int Nz = cd.Nz;
	if ((i < Nz) && (j < Ny) && (k < Nx)) {

        int kill_zwrap = (cd.bc2 != 'P' && i == Nz-1) ? 1 : 0;
		int ip1 = i==Nz-1 ? 0 : i+1;

		int kill_ywrap = (cd.bc1 != 'P' && j == Ny-1) ? 1 : 0;
		int jp1 = j==Ny-1 ? 0 : j+1;

		int kill_xwrap = (cd.bc0 != 'P' && k == Nx-1) ? 1 : 0;
		int kp1 = k==Nx-1 ? 0 : k+1;

		ind_ijk =   (i+0)*Ny*Nx + (j+0)*Nx + (k+0);
		ind_ijp1k = (i+0)*Ny*Nx + (jp1)*Nx + (k+0);
		ind_ip1jk = (ip1)*Ny*Nx + (j+0)*Nx + (k+0);
		ind_ijkp1 = (i+0)*Ny*Nx + (j+0)*Nx + (kp1);

		// compute prefactors
		dt_by_mux = cd.dt/cd.mu_x[ind_ijk].real;
		dt_by_muy = cd.dt/cd.mu_y[ind_ijk].real;
		dt_by_muz = cd.dt/cd.mu_z[ind_ijk].real;

		// Update Hx
		dEzdy = cd.ody * ((kill_ywrap ? 0 : cd.Ez[ind_ijp1k])  - cd.Ez[ind_ijk]);
		dEydz = cd.odz * ((kill_zwrap ? 0 : cd.Ey[ind_ip1jk])  - cd.Ey[ind_ijk]);
		cd.Hx[ind_ijk] += dt_by_mux * (dEydz - dEzdy);

		// update Hy
		dExdz = cd.odz * ((kill_zwrap ? 0 : cd.Ex[ind_ip1jk]) - cd.Ex[ind_ijk]);
		dEzdx = cd.odx * ((kill_xwrap ? 0 : cd.Ez[ind_ijkp1]) - cd.Ez[ind_ijk]);
		cd.Hy[ind_ijk] += dt_by_muy * (dEzdx - dExdz);

		// update Hz
		dEydx = cd.odx * ((kill_xwrap ? 0 : cd.Ey[ind_ijkp1]) - cd.Ey[ind_ijk]);
		dExdy = cd.ody * ((kill_ywrap ? 0 : cd.Ex[ind_ijp1k]) - cd.Ex[ind_ijk]);
		cd.Hz[ind_ijk] += dt_by_muz * (dExdy - dEydx);

		// Do PML updates
		if(k < cd.pml_xmin) {
			// get index in PML array
			ind_pml = i*Ny*(cd.pml_xmin) +j*(cd.pml_xmin) + k;

			// get PML coefficients
			ind_pml_param = cd.pml_xmin - k - 1;
			kappa = cd.kappa_H_x[ind_pml_param];
			b = cd.bHx[ind_pml_param];
			C = cd.cHx[ind_pml_param];

			// Update PML convolution
			cd.pml_Eyx0[ind_pml] = C * dEydx + b*cd.pml_Eyx0[ind_pml];
			cd.pml_Ezx0[ind_pml] = C * dEzdx + b*cd.pml_Ezx0[ind_pml];

			cd.Hz[ind_ijk] -= dt_by_muz * (cd.pml_Eyx0[ind_pml]-dEydx+dEydx/kappa);
			cd.Hy[ind_ijk] += dt_by_muy * (cd.pml_Ezx0[ind_pml]-dEzdx+dEzdx/kappa);

		}
		else if(k  >= cd.pml_xmax) {
			ind_pml = i*Ny*(Nx - cd.pml_xmax) + j*(Nx - cd.pml_xmax) + k - cd.pml_xmax;

			// get pml coefficients
			ind_pml_param = k - cd.pml_xmax + cd.pml_xmin;
			kappa = cd.kappa_H_x[ind_pml_param];
			b = cd.bHx[ind_pml_param];
			C = cd.cHx[ind_pml_param];

			cd.pml_Eyx1[ind_pml] = C * dEydx + b*cd.pml_Eyx1[ind_pml];
			cd.pml_Ezx1[ind_pml] = C * dEzdx + b*cd.pml_Ezx1[ind_pml];

			cd.Hz[ind_ijk] -= dt_by_muz * (cd.pml_Eyx1[ind_pml]-dEydx+dEydx/kappa);
			cd.Hy[ind_ijk] += dt_by_muy * (cd.pml_Ezx1[ind_pml]-dEzdx+dEzdx/kappa);
		}

		if(j < cd.pml_ymin) {
			ind_pml = i*cd.pml_ymin*Nx +j*Nx + k;

			// compute coefficients
			ind_pml_param = cd.pml_ymin - j - 1;
			kappa = cd.kappa_H_y[ind_pml_param];
			b = cd.bHy[ind_pml_param];
			C = cd.cHy[ind_pml_param];

			cd.pml_Exy0[ind_pml] = C * dExdy + b*cd.pml_Exy0[ind_pml];
			cd.pml_Ezy0[ind_pml] = C * dEzdy + b*cd.pml_Ezy0[ind_pml];

			cd.Hz[ind_ijk] += dt_by_muz * (cd.pml_Exy0[ind_pml]-dExdy+dExdy/kappa);
			cd.Hx[ind_ijk] -= dt_by_mux * (cd.pml_Ezy0[ind_pml]-dEzdy+dEzdy/kappa);
		}
		else if(j >= cd.pml_ymax) {
			ind_pml = i*(Ny - cd.pml_ymax)*Nx +(j - cd.pml_ymax)*Nx + k;

			// compute coefficients
			ind_pml_param = j - cd.pml_ymax + cd.pml_ymin;
			kappa = cd.kappa_H_y[ind_pml_param];
			b = cd.bHy[ind_pml_param];
			C = cd.cHy[ind_pml_param];

			cd.pml_Exy1[ind_pml] = C * dExdy + b*cd.pml_Exy1[ind_pml];
			cd.pml_Ezy1[ind_pml] = C * dEzdy + b*cd.pml_Ezy1[ind_pml];

			cd.Hz[ind_ijk] += dt_by_muz * (cd.pml_Exy1[ind_pml]-dExdy+dExdy/kappa);
			cd.Hx[ind_ijk] -= dt_by_mux * (cd.pml_Ezy1[ind_pml]-dEzdy+dEzdy/kappa);
		}

		if(i < cd.pml_zmin) {
			ind_pml = i*Ny*Nx +j*Nx + k;

			// get coefficients
			ind_pml_param = cd.pml_zmin - i - 1;
			kappa = cd.kappa_H_z[ind_pml_param];
			b = cd.bHz[ind_pml_param];
			C = cd.cHz[ind_pml_param];

			cd.pml_Exz0[ind_pml] = C * dExdz + b*cd.pml_Exz0[ind_pml];
			cd.pml_Eyz0[ind_pml] = C * dEydz + b*cd.pml_Eyz0[ind_pml];

			cd.Hx[ind_ijk] += dt_by_mux * (cd.pml_Eyz0[ind_pml]-dEydz+dEydz/kappa);
			cd.Hy[ind_ijk] -= dt_by_muy * (cd.pml_Exz0[ind_pml]-dExdz+dExdz/kappa);
		}
		else if(i > cd.pml_zmax) {
			ind_pml = (i - cd.pml_zmax)*Ny*Nx +j*Nx + k;

			// get coefficients
			ind_pml_param = i - cd.pml_zmax + cd.pml_zmin;
			kappa = cd.kappa_H_z[ind_pml_param];
			b = cd.bHz[ind_pml_param];
			C = cd.cHz[ind_pml_param];

			cd.pml_Exz1[ind_pml] = C * dExdz + b*cd.pml_Exz1[ind_pml];
			cd.pml_Eyz1[ind_pml] = C * dEydz + b*cd.pml_Eyz1[ind_pml];

			cd.Hx[ind_ijk] += dt_by_mux * (cd.pml_Eyz1[ind_pml]-dEydz+dEydz/kappa);
			cd.Hy[ind_ijk] -= dt_by_muy * (cd.pml_Exz1[ind_pml]-dExdz+dExdz/kappa);
		}

	}
}

__global__
void update_H_sources(double t,
					  int i0s, int j0s, int k0s,
					  int Is, int Js, int Ks,
					  complex128 *Mx, complex128 *My, complex128 *Mz)
{
    int ind_ijk, ind_src;
    double src_t;

	int i = blockIdx.z * blockDim.z + threadIdx.z;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i < Is) && (j < Js) && (k < Ks)) {
		ind_ijk = (i+i0s)*cd.Ny*cd.Nx + (j+j0s)*cd.Nx + (k+k0s);
		ind_src = i*Js*Ks + j*Ks + k;

        // update Mx
		src_t = cuda_src_func_t(t, Mx[ind_src].imag);
		cd.Hx[ind_ijk] += src_t * Mx[ind_src].real * cd.dt / cd.mu_x[ind_ijk].real;

        // update My
		src_t = cuda_src_func_t(t, My[ind_src].imag);
		cd.Hy[ind_ijk] += src_t * My[ind_src].real * cd.dt / cd.mu_y[ind_ijk].real;

        // update Mz
		src_t = cuda_src_func_t(t, Mz[ind_src].imag);
		cd.Hz[ind_ijk] += src_t * Mz[ind_src].real * cd.dt / cd.mu_z[ind_ijk].real;
    }
}

void fdtd::FDTD::update_H(double t)
{
	dim3 fields_threadsPerBlock(8, 8, 8);
	dim3 fields_numBlocks(ceil((float)_Nx/fields_threadsPerBlock.x),
						  ceil((float)_Ny/fields_threadsPerBlock.y),
						  ceil((float)_Nz/fields_threadsPerBlock.z));

	update_H_fields <<<fields_numBlocks, fields_threadsPerBlock>>> (t);
    // Update sources
    for(auto const& src : _sources) {
		dim3 sources_threadsPerBlock(8, 8, 8);
		dim3 sources_numBlocks(ceil((float) src.K/sources_threadsPerBlock.x),
							   ceil((float) src.J/sources_threadsPerBlock.y),
							   ceil((float) src.I/sources_threadsPerBlock.z));

		update_H_sources <<<sources_numBlocks, sources_threadsPerBlock>>>
			(t,
			 src.i0, src.j0, src.k0,
			 src.I, src.J, src.K,
			 src.Mx, src.My, src.Mz);
	}
}

enum action { ACTION_NOP, ACTION_ZERO, ACTION_FLIP, ACTION_COPY };

__global__
void update_E_fields(double t)
{
	double b_x, b_y, b_z;

    int ind_ijk, ind_im1jk, ind_ijm1k, ind_ijkm1, ind_pml, ind_pml_param;

    double dHxdy, dHxdz, dHydx, dHydz, dHzdx, dHzdy;

    double b, C, kappa;

	int i = blockIdx.z * blockDim.z + threadIdx.z;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i < cd.Nz) && (j < cd.Ny) && (k < cd.Nx)) {
        int action_zwrap = (cd.bc2 == 'P' || i != 0 ? ACTION_NOP :
							cd.bc2 == '0'           ? ACTION_ZERO :
							cd.bc2 == 'E'           ? ACTION_FLIP : ACTION_COPY);
		int im1 = i==0 ? cd.Nz-1 : i-1;
		int action_ywrap = (cd.bc1 == 'P' || j != 0 ? ACTION_NOP :
							cd.bc1 == '0'           ? ACTION_ZERO :
							cd.bc1 == 'E'           ? ACTION_FLIP : ACTION_COPY);
		int jm1 = j==0 ? cd.Ny-1 : j-1;
		int action_xwrap = (cd.bc0 == 'P' || k != 0 ? ACTION_NOP :
							cd.bc0 == '0'           ? ACTION_ZERO :
							cd.bc0 == 'E'           ? ACTION_FLIP : ACTION_COPY);
		int km1 = k==0 ? cd.Nx-1 : k-1;

		ind_ijk   = (i-0)*cd.Ny*cd.Nx + (j-0)*cd.Nx + (k-0);
		ind_ijm1k = (i-0)*cd.Ny*cd.Nx + (jm1)*cd.Nx + (k-0);
		ind_im1jk = (im1)*cd.Ny*cd.Nx + (j-0)*cd.Nx + (k-0);
		ind_ijkm1 = (i-0)*cd.Ny*cd.Nx + (j-0)*cd.Nx + (km1);

		b_x = cd.dt/cd.eps_x[ind_ijk].real;
		b_y = cd.dt/cd.eps_y[ind_ijk].real;
		b_z = cd.dt/cd.eps_z[ind_ijk].real;

		// Update Ex
		dHzdy = cd.ody*(cd.Hz[ind_ijk] - (action_ywrap == ACTION_ZERO ? 0.0 :
										  action_ywrap == ACTION_COPY ?  cd.Hz[ind_ijk] :
										  action_ywrap == ACTION_FLIP ? -cd.Hz[ind_ijk] : cd.Hz[ind_ijm1k]));
		dHydz = cd.odz*(cd.Hy[ind_ijk] - (action_zwrap == ACTION_ZERO ? 0.0 :
										  action_zwrap == ACTION_COPY ?  cd.Hy[ind_ijk] :
										  action_zwrap == ACTION_FLIP ? -cd.Hy[ind_ijk] : cd.Hy[ind_im1jk]));
		cd.Ex[ind_ijk] += (dHzdy - dHydz) * b_x;

		// Update Ey
		dHxdz = cd.odz*(cd.Hx[ind_ijk] - (action_zwrap == ACTION_ZERO ? 0.0 :
									   action_zwrap == ACTION_COPY ?  cd.Hx[ind_ijk] :
									   action_zwrap == ACTION_FLIP ? -cd.Hx[ind_ijk] : cd.Hx[ind_im1jk]));
		dHzdx = cd.odx*(cd.Hz[ind_ijk] - (action_xwrap == ACTION_ZERO ? 0.0 :
									   action_xwrap == ACTION_COPY ?  cd.Hz[ind_ijk] :
									   action_xwrap == ACTION_FLIP ? -cd.Hz[ind_ijk] : cd.Hz[ind_ijkm1]));
		cd.Ey[ind_ijk] += (dHxdz - dHzdx) * b_y;

		// Update Ez
		dHydx = cd.odx*(cd.Hy[ind_ijk] - (action_xwrap == ACTION_ZERO ? 0.0 :
									   action_xwrap == ACTION_COPY ?  cd.Hy[ind_ijk] :
									   action_xwrap == ACTION_FLIP ? -cd.Hy[ind_ijk] : cd.Hy[ind_ijkm1]));
		dHxdy = cd.ody*(cd.Hx[ind_ijk] - (action_ywrap == ACTION_ZERO ? 0.0 :
									   action_ywrap == ACTION_COPY ?  cd.Hx[ind_ijk] :
									   action_ywrap == ACTION_FLIP ? -cd.Hx[ind_ijk] : cd.Hx[ind_ijm1k]));
		cd.Ez[ind_ijk] += (dHydx - dHxdy) * b_z;

		// Do PML updates
		if(k < cd.pml_xmin) {
			ind_pml = i*cd.Ny*(cd.pml_xmin) +j*(cd.pml_xmin) + k;

			// get PML coefficients
			ind_pml_param = cd.pml_xmin - k - 1;
			kappa = cd.kappa_E_x[ind_pml_param];
			b = cd.bEx[ind_pml_param];
			C = cd.cEx[ind_pml_param];

			cd.pml_Hyx0[ind_pml] = C * dHydx + b*cd.pml_Hyx0[ind_pml];
			cd.pml_Hzx0[ind_pml] = C * dHzdx + b*cd.pml_Hzx0[ind_pml];

			cd.Ez[ind_ijk] += (cd.pml_Hyx0[ind_pml]-dHydx+dHydx/kappa) * b_z;
			cd.Ey[ind_ijk] -= (cd.pml_Hzx0[ind_pml]-dHzdx+dHzdx/kappa) * b_y;

		}
		else if(k >= cd.pml_xmax) {
			ind_pml = i*cd.Ny*(cd.Nx - cd.pml_xmax) +j*(cd.Nx - cd.pml_xmax) + k - cd.pml_xmax;

			// get coefficients
			ind_pml_param = k - cd.pml_xmax + cd.pml_xmin;
			kappa = cd.kappa_E_x[ind_pml_param];
			b = cd.bEx[ind_pml_param];
			C = cd.cEx[ind_pml_param];

			cd.pml_Hyx1[ind_pml] = C * dHydx + b*cd.pml_Hyx1[ind_pml];
			cd.pml_Hzx1[ind_pml] = C * dHzdx + b*cd.pml_Hzx1[ind_pml];

			cd.Ez[ind_ijk] += (cd.pml_Hyx1[ind_pml]-dHydx+dHydx/kappa) * b_z;
			cd.Ey[ind_ijk] -= (cd.pml_Hzx1[ind_pml]-dHzdx+dHzdx/kappa) * b_y;
		}

		if(j < cd.pml_ymin) {
			ind_pml = i*cd.pml_ymin*cd.Nx +j*cd.Nx + k;

			// get coefficients
			ind_pml_param = cd.pml_ymin - j - 1;
			kappa = cd.kappa_E_y[ind_pml_param];
			b = cd.bEy[ind_pml_param];
			C = cd.cEy[ind_pml_param];

			cd.pml_Hxy0[ind_pml] = C * dHxdy + b*cd.pml_Hxy0[ind_pml];
			cd.pml_Hzy0[ind_pml] = C * dHzdy + b*cd.pml_Hzy0[ind_pml];

			cd.Ez[ind_ijk] -= (cd.pml_Hxy0[ind_pml]-dHxdy+dHxdy/kappa) * b_z;
			cd.Ex[ind_ijk] += (cd.pml_Hzy0[ind_pml]-dHzdy+dHzdy/kappa) * b_x;
		}
		else if(j >= cd.pml_ymax) {
			ind_pml = i*(cd.Ny - cd.pml_ymax)*cd.Nx +(j - cd.pml_ymax)*cd.Nx + k;

			// get coefficients
			ind_pml_param = j - cd.pml_ymax + cd.pml_ymin;
			kappa = cd.kappa_E_y[ind_pml_param];
			b = cd.bEy[ind_pml_param];
			C = cd.cEy[ind_pml_param];

			cd.pml_Hxy1[ind_pml] = C * dHxdy + b*cd.pml_Hxy1[ind_pml];
			cd.pml_Hzy1[ind_pml] = C * dHzdy + b*cd.pml_Hzy1[ind_pml];

			cd.Ez[ind_ijk] -= (cd.pml_Hxy1[ind_pml]-dHxdy+dHxdy/kappa) * b_z;
			cd.Ex[ind_ijk] += (cd.pml_Hzy1[ind_pml]-dHzdy+dHzdy/kappa) * b_x;
		}

		if(i < cd.pml_zmin) {
			ind_pml = i*cd.Ny*cd.Nx +j*cd.Nx + k;

			// get coefficients
			ind_pml_param = cd.pml_zmin - i - 1;
			kappa = cd.kappa_E_z[ind_pml_param];
			b = cd.bEz[ind_pml_param];
			C = cd.cEz[ind_pml_param];

			cd.pml_Hxz0[ind_pml] = C * dHxdz + b*cd.pml_Hxz0[ind_pml];
			cd.pml_Hyz0[ind_pml] = C * dHydz + b*cd.pml_Hyz0[ind_pml];

			cd.Ex[ind_ijk] -= (cd.pml_Hyz0[ind_pml]-dHydz+dHydz/kappa) * b_x;
			cd.Ey[ind_ijk] += (cd.pml_Hxz0[ind_pml]-dHxdz+dHxdz/kappa) * b_y;
		}
		else if(i > cd.pml_zmax) {
			ind_pml = (i - cd.pml_zmax)*cd.Ny*cd.Nx +j*cd.Nx + k;

			// compute coefficients
			ind_pml_param = i - cd.pml_zmax + cd.pml_zmin;
			kappa = cd.kappa_E_z[ind_pml_param];
			b = cd.bEz[ind_pml_param];
			C = cd.cEz[ind_pml_param];

			cd.pml_Hxz1[ind_pml] = C * dHxdz + b*cd.pml_Hxz1[ind_pml];
			cd.pml_Hyz1[ind_pml] = C * dHydz + b*cd.pml_Hyz1[ind_pml];

			cd.Ex[ind_ijk] -= (cd.pml_Hyz1[ind_pml]-dHydz+dHydz/kappa) * b_x;
			cd.Ey[ind_ijk] += (cd.pml_Hxz1[ind_pml]-dHxdz+dHxdz/kappa) * b_y;
		}
	}
}

__global__
void update_E_sources(double t,
					  int i0s, int j0s, int k0s,
					  int Is, int Js, int Ks,
					  complex128 *Jx, complex128 *Jy, complex128 *Jz)
{
    int ind_ijk, ind_src;
    double src_t;
	double b;

	int i = blockIdx.z * blockDim.z + threadIdx.z;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if ((i < Is) && (j < Js) && (k < Ks)) {

		ind_ijk = (i+i0s)*cd.Ny*cd.Nx + (j+j0s)*cd.Nx + (k+k0s);
		ind_src = i*Js*Ks + j*Ks + k;

		// update Jx
		b = cd.dt/cd.eps_x[ind_ijk].real;
		src_t = cuda_src_func_t(t, Jx[ind_src].imag);
		cd.Ex[ind_ijk] -= src_t * Jx[ind_src].real * b;

		// update Jy
		b = cd.dt/cd.eps_y[ind_ijk].real;
		src_t = cuda_src_func_t(t, Jy[ind_src].imag);
		cd.Ey[ind_ijk] -= src_t * Jy[ind_src].real * b;

		// update Jz
		b = cd.dt/cd.eps_z[ind_ijk].real;
		src_t = cuda_src_func_t(t, Jz[ind_src].imag);
		cd.Ez[ind_ijk] -= src_t * Jz[ind_src].real * b;
	}
}

void fdtd::FDTD::update_E(double t)
{
	dim3 fields_threadsPerBlock(8, 8, 8);
	dim3 fields_numBlocks(ceil((float)_Nx/fields_threadsPerBlock.x),
						  ceil((float)_Ny/fields_threadsPerBlock.y),
						  ceil((float)_Nz/fields_threadsPerBlock.z));

	update_E_fields <<<fields_numBlocks, fields_threadsPerBlock>>> (t);

    // Update sources
    for(auto const& src : _sources) {
		dim3 sources_threadsPerBlock(8, 8, 8);
		dim3 sources_numBlocks(ceil((float) src.K/sources_threadsPerBlock.x),
							   ceil((float) src.J/sources_threadsPerBlock.y),
							   ceil((float) src.I/sources_threadsPerBlock.z));

		update_E_sources <<<sources_numBlocks, sources_threadsPerBlock>>>
			(t,
			 src.i0, src.j0, src.k0,
			 src.I, src.J, src.K,
			 src.Jx, src.Jy, src.Jz);
	}
}


void fdtd::FDTD::update(double start_time, int num_time_steps)
{
    _hcd.odx = _R/_dx;
	_hcd.ody = _R/_dy;
	_hcd.odz = _R/_dz;

	_hcd.Nx = _Nx;
	_hcd.Ny = _Ny;
	_hcd.Nz = _Nz;

    _hcd.pml_xmin = _w_pml_x0;
	_hcd.pml_xmax = _Nx-_w_pml_x1;
	_hcd.pml_ymin = _w_pml_y0;
	_hcd.pml_ymax = _Ny-_w_pml_y1;
	_hcd.pml_zmin = _w_pml_z0;
	_hcd.pml_zmax = _Nz-_w_pml_z1;

	hipMemcpyToSymbol(HIP_SYMBOL(&_hcd), &cd, sizeof(CudaData));

	double time = start_time;
    for(int i = 0; i < num_time_steps; ++i) {
		update_H(time);
		update_E(time + _dt/2.0);
		time += _dt;
	}
	hipDeviceSynchronize();
}


///////////////////////////////////////////////////////////////////////////
// PML Management
///////////////////////////////////////////////////////////////////////////


void fdtd::FDTD::set_pml_widths(int xmin, int xmax, int ymin, int ymax, int zmin, int zmax)
{
    _w_pml_x0 = xmin; _w_pml_x1 = xmax;
    _w_pml_y0 = ymin; _w_pml_y1 = ymax;
    _w_pml_z0 = zmin; _w_pml_z1 = zmax;
}

void fdtd::FDTD::set_pml_properties(double sigma, double alpha, double kappa, double pow)
{
    _sigma = sigma;
    _alpha = alpha;
    _kappa = kappa;
    _pow   = pow;

    compute_pml_params();
}

void fdtd::FDTD::build_pml()
{
    int N,
        xmin = _w_pml_x0, xmax = _Nx-_w_pml_x1,
        ymin = _w_pml_y0, ymax = _Ny-_w_pml_y1,
        zmin = _w_pml_z0, zmax = _Nz-_w_pml_z1;

    // touches xmin boudary
    if(0 < xmin) {
        N = _Nz * _Ny * xmin;

        // Clean up old arrays and allocate new ones
        hipFree(_hcd.pml_Eyx0);
        hipFree(_hcd.pml_Ezx0);
        hipMallocManaged((void **)&_hcd.pml_Eyx0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Ezx0, N*sizeof(double));

        hipFree(_hcd.pml_Hyx0);
        hipFree(_hcd.pml_Hzx0);
        hipMallocManaged((void **)&_hcd.pml_Hyx0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hzx0, N*sizeof(double));
    }

    // touches xmax boundary
    if(_Nx > xmax) {
        N = _Nz * _Ny * (_Nx - xmax);

        // Clean up old arrays and allocate new ones
        hipFree(_hcd.pml_Eyx1);
        hipFree(_hcd.pml_Ezx1);
        hipMallocManaged((void **)&_hcd.pml_Eyx1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Ezx1, N*sizeof(double));

        hipFree(_hcd.pml_Hyx1);
        hipFree(_hcd.pml_Hzx1);
        hipMallocManaged((void **)&_hcd.pml_Hyx1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hzx1, N*sizeof(double));
    }

    // touches ymin boundary
    if(0 < ymin) {
        N = _Nz * _Nx * ymin;

        hipFree(_hcd.pml_Exy0);
		hipFree(_hcd.pml_Ezy0);
        hipMallocManaged((void **)&_hcd.pml_Exy0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Ezy0, N*sizeof(double));

        hipFree(_hcd.pml_Hxy0);
        hipFree(_hcd.pml_Hzy0);
        hipMallocManaged((void **)&_hcd.pml_Hxy0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hzy0, N*sizeof(double));
    }

    // touches ymax boundary
    if(_Ny > ymax) {
        N = _Nz * _Nx * (_Ny - ymax);

        hipFree(_hcd.pml_Exy1);
        hipFree(_hcd.pml_Ezy1);
        hipMallocManaged((void **)&_hcd.pml_Exy1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Ezy1, N*sizeof(double));

        hipFree(_hcd.pml_Hxy1);
		hipFree(_hcd.pml_Hzy1);
        hipMallocManaged((void **)&_hcd.pml_Hxy1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hzy1, N*sizeof(double));
    }

    // touches zmin boundary
    if(0 < zmin) {
        N = _Ny * _Nx * zmin;

        hipFree(_hcd.pml_Exz0);
        hipFree(_hcd.pml_Eyz0);
        hipMallocManaged((void **)&_hcd.pml_Exz0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Eyz0, N*sizeof(double));

        hipFree(_hcd.pml_Hxz0);
        hipFree(_hcd.pml_Hyz0);
        hipMallocManaged((void **)&_hcd.pml_Hxz0, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hyz0, N*sizeof(double));
    }

    // touches zmax boundary
    if(_Nz > zmax) {
        N = _Ny * _Nx * (_Nz - zmax);

        hipFree(_hcd.pml_Exz1);
        hipFree(_hcd.pml_Eyz1);
        hipMallocManaged((void **)&_hcd.pml_Exz1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Eyz1, N*sizeof(double));

        hipFree(_hcd.pml_Hxz1);
		hipFree(_hcd.pml_Hyz1);
        hipMallocManaged((void **)&_hcd.pml_Hxz1, N*sizeof(double));
        hipMallocManaged((void **)&_hcd.pml_Hyz1, N*sizeof(double));
    }

    // (re)compute the spatially-dependent PML parameters
    compute_pml_params();
}

void fdtd::FDTD::reset_pml()
{
    int N,
        xmin = _w_pml_x0, xmax = _Nx-_w_pml_x1,
        ymin = _w_pml_y0, ymax = _Ny-_w_pml_y1,
        zmin = _w_pml_z0, zmax = _Nz-_w_pml_z1;

    // touches xmin boudary
    if(0 < xmin) {
        N = _Nz * _Ny * xmin;
        std::fill(_hcd.pml_Eyx0, _hcd.pml_Eyx0 + N, 0);
        std::fill(_hcd.pml_Ezx0, _hcd.pml_Ezx0 + N, 0);
        std::fill(_hcd.pml_Hyx0, _hcd.pml_Hyx0 + N, 0);
        std::fill(_hcd.pml_Hzx0, _hcd.pml_Hzx0 + N, 0);
    }

    // touches xmax boundary
    if(0 +_Nx > xmax) {
        N = _Nz * _Ny * (_Nx - xmax);

        std::fill(_hcd.pml_Eyx1, _hcd.pml_Eyx1 + N, 0);
        std::fill(_hcd.pml_Ezx1, _hcd.pml_Ezx1 + N, 0);
        std::fill(_hcd.pml_Hyx1, _hcd.pml_Hyx1 + N, 0);
        std::fill(_hcd.pml_Hzx1, _hcd.pml_Hzx1 + N, 0);
    }

    // touches ymin boundary
    if(0 < ymin) {
        N = _Nz * _Nx * ymin;

        std::fill(_hcd.pml_Exy0, _hcd.pml_Exy0 + N, 0);
        std::fill(_hcd.pml_Ezy0, _hcd.pml_Ezy0 + N, 0);
        std::fill(_hcd.pml_Hxy0, _hcd.pml_Hxy0 + N, 0);
        std::fill(_hcd.pml_Hzy0, _hcd.pml_Hzy0 + N, 0);
    }

    // touches ymax boundary
    if(_Ny > ymax) {
        N = _Nz * _Nx * (_Ny - ymax);

        std::fill(_hcd.pml_Exy1, _hcd.pml_Exy1 + N, 0);
        std::fill(_hcd.pml_Ezy1, _hcd.pml_Ezy1 + N, 0);
        std::fill(_hcd.pml_Hxy1, _hcd.pml_Hxy1 + N, 0);
        std::fill(_hcd.pml_Hzy1, _hcd.pml_Hzy1 + N, 0);
    }

    // touches zmin boundary
    if(0 < zmin) {
        N = _Ny * _Nx * zmin;

        std::fill(_hcd.pml_Exz0, _hcd.pml_Exz0 + N, 0);
        std::fill(_hcd.pml_Eyz0, _hcd.pml_Eyz0 + N, 0);
        std::fill(_hcd.pml_Hxz0, _hcd.pml_Hxz0 + N, 0);
        std::fill(_hcd.pml_Hyz0, _hcd.pml_Hyz0 + N, 0);
    }

    // touches zmax boundary
    if(_Nz > zmax) {
        N = _Ny * _Nx * (_Nz - zmax);

        std::fill(_hcd.pml_Exz1, _hcd.pml_Exz1 + N, 0);
        std::fill(_hcd.pml_Eyz1, _hcd.pml_Eyz1 + N, 0);
        std::fill(_hcd.pml_Hxz1, _hcd.pml_Hxz1 + N, 0);
        std::fill(_hcd.pml_Hyz1, _hcd.pml_Hyz1 + N, 0);
    }

}

void fdtd::FDTD::compute_pml_params()
{
    double pml_dist, pml_factor, sigma, alpha, kappa, b, c;

    // clean up the previous arrays and allocate new ones
    hipFree(_hcd.kappa_H_x);
	hipMallocManaged((void **)&_hcd.kappa_H_x, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.kappa_H_y);
	hipMallocManaged((void **)&_hcd.kappa_H_y, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.kappa_H_z);
	hipMallocManaged((void **)&_hcd.kappa_H_z, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

    hipFree(_hcd.kappa_E_x);
	hipMallocManaged((void **)&_hcd.kappa_E_x, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.kappa_E_y);
	hipMallocManaged((void **)&_hcd.kappa_E_y, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.kappa_E_z);
	hipMallocManaged((void **)&_hcd.kappa_E_z, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

    hipFree(_hcd.bHx);
	hipMallocManaged((void **)&_hcd.bHx, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.bHy);
	hipMallocManaged((void **)&_hcd.bHy, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.bHz);
	hipMallocManaged((void **)&_hcd.bHz, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

    hipFree(_hcd.bEx);
	hipMallocManaged((void **)&_hcd.bEx, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.bEy);
	hipMallocManaged((void **)&_hcd.bEy, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.bEz);
	hipMallocManaged((void **)&_hcd.bEz, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

    hipFree(_hcd.cHx);
	hipMallocManaged((void **)&_hcd.cHx, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.cHy);
	hipMallocManaged((void **)&_hcd.cHy, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.cHz);
	hipMallocManaged((void **)&_hcd.cHz, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

    hipFree(_hcd.cEx);
	hipMallocManaged((void **)&_hcd.cEx, sizeof(double)*(_w_pml_x0 + _w_pml_x1));
    hipFree(_hcd.cEy);
	hipMallocManaged((void **)&_hcd.cEy, sizeof(double)*(_w_pml_y0 + _w_pml_y1));
    hipFree(_hcd.cEz);
	hipMallocManaged((void **)&_hcd.cEz, sizeof(double)*(_w_pml_z0 + _w_pml_z1));

	std::cout << "after malloc, _hcd.cEz: " << _hcd.cEz << std::endl;	

    // calculate the PML parameters. These parameters are all functions of
    // the distance from the ONSET of the PML edge (which begins in the simulation
    // domain interior.
    // Note: PML parameters are ordered such that distance from PML onset
    // always increases with index.

    // setup xmin PML parameters
    for(int k = 0; k < _w_pml_x0; k++) {
        pml_dist = double(k - 0.5)/_w_pml_x0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);
        if(pml_factor < 0) pml_factor = 0;

        // compute H coefficients
        sigma = _sigma * pml_factor;
        alpha = _alpha * (1-pml_factor);
        kappa = (_kappa-1.0) * pml_factor+1.0;
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_x[k] = kappa;
        _hcd.bHx[k] = b;
        _hcd.cHx[k] = c;

        pml_dist = double(k)/_w_pml_x0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        // compute E coefficients
        sigma = _sigma * pml_factor;
        alpha = _alpha * (1-pml_factor);
        kappa = (_kappa-1) * pml_factor+1;
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_x[k] = kappa;
        _hcd.bEx[k] = b;
        _hcd.cEx[k] = c;

    }
    for(int k = 0; k < _w_pml_x1; k++) {
        // compute H coefficients
        pml_dist = double(k + 0.5)/_w_pml_x1; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_x[_w_pml_x0 + k] = kappa;
        _hcd.bHx[_w_pml_x0 + k] = b;
        _hcd.cHx[_w_pml_x0 + k] = c;

        //compute E coefficients
        pml_dist = double(k)/_w_pml_x1; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_x[_w_pml_x0 + k] = kappa;
        _hcd.bEx[_w_pml_x0 + k] = b;
        _hcd.cEx[_w_pml_x0 + k] = c;
    }
    for(int j = 0; j < _w_pml_y0; j++) {
        // calc H coefficients
        pml_dist = double(j - 0.5)/_w_pml_y0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);
        if(pml_factor < 0) pml_factor = 0;

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_y[j] = kappa;
        _hcd.bHy[j] = b;
        _hcd.cHy[j] = c;

        // calc E coefficients
        pml_dist = double(j)/_w_pml_y0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_y[j] = kappa;
        _hcd.bEy[j] = b;
        _hcd.cEy[j] = c;

    }
    for(int j = 0; j < _w_pml_y1; j++) {
         // calc H coeffs
         pml_dist = double(j + 0.5)/_w_pml_y1; // distance from pml edge
         pml_factor = pml_ramp(pml_dist);

         sigma = _sigma * pml_factor;
         kappa = (_kappa-1) * pml_factor+1;
         alpha = _alpha * (1-pml_factor);
         b = exp(-_dt*(sigma/kappa + alpha));
         if(b == 1) c = 0;
         else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_y[_w_pml_y0 + j] = kappa;
        _hcd.bHy[_w_pml_y0 + j] = b;
        _hcd.cHy[_w_pml_y0 + j] = c;

        // compute E coefficients
        pml_dist = double(j)/_w_pml_y1; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_y[_w_pml_y0 + j] = kappa;
        _hcd.bEy[_w_pml_y0 + j] = b;
        _hcd.cEy[_w_pml_y0 + j] = c;
    }

    for(int i = 0; i < _w_pml_z0; i++) {
        // calc H coeffs
        pml_dist = double(i)/_w_pml_z0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c= 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_z[i] = kappa;
        _hcd.bHz[i] = b;
        _hcd.cHz[i] = c;

        // calc E coeffs
        pml_dist = double(i+0.5)/_w_pml_z0; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        // compute coefficients
        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_z[i] = kappa;
        _hcd.bEz[i] = b;
        _hcd.cEz[i] = c;
    }

    for(int i = 0; i < _w_pml_z1; i++) {
        // calc H coeffs
        pml_dist = double(i)/_w_pml_z1; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);

        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_H_z[_w_pml_z0 + i] = kappa;
        _hcd.bHz[_w_pml_z0 + i] = b;
        _hcd.cHz[_w_pml_z0 + i] = c;

        // calc E coeffs
        pml_dist = double(i - 0.5)/_w_pml_z1; // distance from pml edge
        pml_factor = pml_ramp(pml_dist);
        if(pml_factor < 0) pml_factor = 0;

        // compute coefficients
        sigma = _sigma * pml_factor;
        kappa = (_kappa-1) * pml_factor+1;
        alpha = _alpha * (1-pml_factor);
        b = exp(-_dt*(sigma/kappa + alpha));
        if(b == 1) c = 0;
        else c = (b - 1)*sigma / (sigma*kappa + kappa*kappa*alpha);

        _hcd.kappa_E_z[_w_pml_z0 + i] = kappa;
        _hcd.bEz[_w_pml_z0 + i] = b;
        _hcd.cEz[_w_pml_z0 + i] = c;
    }
}

double fdtd::FDTD::pml_ramp(double pml_dist)
{
    return std::pow(pml_dist, _pow);
}

///////////////////////////////////////////////////////////////////////////
// Amp/Phase Calculation management Management
///////////////////////////////////////////////////////////////////////////
void fdtd::FDTD::set_t0_arrays(complex128 *Ex_t0, complex128 *Ey_t0, complex128 *Ez_t0,
                                complex128 *Hx_t0, complex128 *Hy_t0, complex128 *Hz_t0)
{
    _Ex_t0 = Ex_t0; _Ey_t0 = Ey_t0; _Ez_t0 = Ez_t0;
    _Hx_t0 = Hx_t0; _Hy_t0 = Hy_t0; _Hz_t0 = Hz_t0;
}

void fdtd::FDTD::set_t1_arrays(complex128 *Ex_t1, complex128 *Ey_t1, complex128 *Ez_t1,
complex128 *Hx_t1, complex128 *Hy_t1, complex128 *Hz_t1)
{
_Ex_t1 = Ex_t1; _Ey_t1 = Ey_t1; _Ez_t1 = Ez_t1;
_Hx_t1 = Hx_t1; _Hy_t1 = Hy_t1; _Hz_t1 = Hz_t1;
}

void fdtd::FDTD::capture_t0_fields()
{
    int ind_ijk;

    for(int i = 0; i < _Nz; i++) {
        for(int j = 0; j < _Ny; j++) {
            for(int k = 0; k < _Nx; k++) {
                ind_ijk = (i)*(_Ny)*(_Nx) + (j)*(_Nx) + k;

                // Copy the fields at the current time to the auxillary arrays
                _Ex_t0[ind_ijk] = _hcd.Ex[ind_ijk];
                _Ey_t0[ind_ijk] = _hcd.Ey[ind_ijk];
                _Ez_t0[ind_ijk] = _hcd.Ez[ind_ijk];

                _Hx_t0[ind_ijk] = _hcd.Hx[ind_ijk];
                _Hy_t0[ind_ijk] = _hcd.Hy[ind_ijk];
                _Hz_t0[ind_ijk] = _hcd.Hz[ind_ijk];
            }
        }
    }

}

void fdtd::FDTD::capture_t1_fields()
{
    int ind_ijk;

    for(int i = 0; i < _Nz; i++) {
        for(int j = 0; j < _Ny; j++) {
            for(int k = 0; k < _Nx; k++) {
                ind_ijk = (i)*(_Ny)*(_Nx) + (j)*(_Nx) + k;

                // Copy the fields at the current time to the auxillary arrays
                _Ex_t1[ind_ijk] = _hcd.Ex[ind_ijk];
                _Ey_t1[ind_ijk] = _hcd.Ey[ind_ijk];
                _Ez_t1[ind_ijk] = _hcd.Ez[ind_ijk];

                _Hx_t1[ind_ijk] = _hcd.Hx[ind_ijk];
                _Hy_t1[ind_ijk] = _hcd.Hy[ind_ijk];
                _Hz_t1[ind_ijk] = _hcd.Hz[ind_ijk];
            }
        }
    }

}

void fdtd::FDTD::calc_complex_fields(double t0, double t1)
{
    double f0, f1, phi, A, t0H, t1H;
    int ind_ijk;

    t0H = t0 - 0.5*_dt;
    t1H = t1 - 0.5*_dt;

    for(int i = 0; i < _Nz; i++) {
        for(int j = 0; j < _Ny; j++) {
            for(int k = 0; k < _Nx; k++) {
                ind_ijk = (i)*(_Ny)*(_Nx) + (j)*(_Nx) + k;

                // Compute amplitude and phase for Ex
                // Note: we are careful to assume exp(-i*w*t) time dependence
                f0 = _Ex_t0[ind_ijk].real;
                f1 = _hcd.Ex[ind_ijk];
                phi = calc_phase(t0, t1, f0, f1);
                A = calc_amplitude(t0, t1, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ex_t0[ind_ijk].real = A*cos(phi);
                _Ex_t0[ind_ijk].imag = -A*sin(phi);

                // Ey
                f0 = _Ey_t0[ind_ijk].real;
                f1 = _hcd.Ey[ind_ijk];
                phi = calc_phase(t0, t1, f0, f1);
                A = calc_amplitude(t0, t1, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ey_t0[ind_ijk].real = A*cos(phi);
                _Ey_t0[ind_ijk].imag = -A*sin(phi);

                // Ez
                f0 = _Ez_t0[ind_ijk].real;
                f1 = _hcd.Ez[ind_ijk];
                phi = calc_phase(t0, t1, f0, f1);
                A = calc_amplitude(t0, t1, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ez_t0[ind_ijk].real = A*cos(phi);
                _Ez_t0[ind_ijk].imag = -A*sin(phi);

                // Hx
                f0 = _Hx_t0[ind_ijk].real;
                f1 = _hcd.Hx[ind_ijk];
                phi = calc_phase(t0H, t1H, f0, f1);
                A = calc_amplitude(t0H, t1H, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hx_t0[ind_ijk].real = A*cos(phi);
                _Hx_t0[ind_ijk].imag = -A*sin(phi);

                // Hy
                f0 = _Hy_t0[ind_ijk].real;
                f1 = _hcd.Hy[ind_ijk];
                phi = calc_phase(t0H, t1H, f0, f1);
                A = calc_amplitude(t0H, t1H, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hy_t0[ind_ijk].real = A*cos(phi);
                _Hy_t0[ind_ijk].imag = -A*sin(phi);

                // Hz
                f0 = _Hz_t0[ind_ijk].real;
                f1 = _hcd.Hz[ind_ijk];
                phi = calc_phase(t0H, t1H, f0, f1);
                A = calc_amplitude(t0H, t1H, f0, f1, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hz_t0[ind_ijk].real = A*cos(phi);
                _Hz_t0[ind_ijk].imag = -A*sin(phi);
            }
        }
    }

}


void fdtd::FDTD::calc_complex_fields(double t0, double t1, double t2)
{
    double f0, f1, f2, phi, A, t0H, t1H, t2H;
    int ind_ijk;

    t0H = t0 - 0.5*_dt;
    t1H = t1 - 0.5*_dt;
    t2H = t2 - 0.5*_dt;

    for(int i = 0; i < _Nz; i++) {
        for(int j = 0; j < _Ny; j++) {
            for(int k = 0; k < _Nx; k++) {
                ind_ijk = (i)*(_Ny)*(_Nx) + (j)*(_Nx) + k;

                // Compute amplitude and phase for Ex
                // Note: we are careful to assume exp(-i*w*t) time dependence
                f0 = _Ex_t0[ind_ijk].real;
                f1 = _Ex_t1[ind_ijk].real;
                f2 = _hcd.Ex[ind_ijk];
                phi = calc_phase(t0, t1, t2, f0, f1, f2);
                A = calc_amplitude(t0, t1, t2, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ex_t0[ind_ijk].real = A*cos(phi);
                _Ex_t0[ind_ijk].imag = -A*sin(phi);

                // Ey
                f0 = _Ey_t0[ind_ijk].real;
                f1 = _Ey_t1[ind_ijk].real;
                f2 = _hcd.Ey[ind_ijk];
                phi = calc_phase(t0, t1, t2, f0, f1, f2);
                A = calc_amplitude(t0, t1, t2, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ey_t0[ind_ijk].real = A*cos(phi);
                _Ey_t0[ind_ijk].imag = -A*sin(phi);

                // Ez
                f0 = _Ez_t0[ind_ijk].real;
                f1 = _Ez_t1[ind_ijk].real;
                f2 = _hcd.Ez[ind_ijk];
                phi = calc_phase(t0, t1, t2, f0, f1, f2);
                A = calc_amplitude(t0, t1, t2, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Ez_t0[ind_ijk].real = A*cos(phi);
                _Ez_t0[ind_ijk].imag = -A*sin(phi);

                // Hx
                f0 = _Hx_t0[ind_ijk].real;
                f1 = _Hx_t1[ind_ijk].real;
                f2 = _hcd.Hx[ind_ijk];
                phi = calc_phase(t0H, t1H, t2H, f0, f1, f2);
                A = calc_amplitude(t0H, t1H, t2H, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hx_t0[ind_ijk].real = A*cos(phi);
                _Hx_t0[ind_ijk].imag = -A*sin(phi);

                // Hy
                f0 = _Hy_t0[ind_ijk].real;
                f1 = _Hy_t1[ind_ijk].real;
                f2 = _hcd.Hy[ind_ijk];
                phi = calc_phase(t0H, t1H, t2H, f0, f1, f2);
                A = calc_amplitude(t0H, t1H, t2H, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hy_t0[ind_ijk].real = A*cos(phi);
                _Hy_t0[ind_ijk].imag = -A*sin(phi);

                // Hz
                f0 = _Hz_t0[ind_ijk].real;
                f1 = _Hz_t1[ind_ijk].real;
                f2 = _hcd.Hz[ind_ijk];
                phi = calc_phase(t0H, t1H, t2H, f0, f1, f2);
                A = calc_amplitude(t0H, t1H, t2H, f0, f1, f2, phi);
                if(A < 0) {
                    A *= -1;
                    phi += M_PI;
                }
                _Hz_t0[ind_ijk].real = A*cos(phi);
                _Hz_t0[ind_ijk].imag = -A*sin(phi);

            }
        }
    }
}

inline double fdtd::calc_phase(double t0, double t1, double f0, double f1)
{
    if(f0 == 0.0 and f1 == 0) {
        return 0.0;
    }
    else {
        return atan((f1*sin(t0)-f0*sin(t1))/(f0*cos(t1)-f1*cos(t0)));
    }
}

inline double fdtd::calc_amplitude(double t0, double t1, double f0, double f1, double phase)
{
    if(f0*f0 > f1*f1) {
        return f1 / (sin(t1)*cos(phase) + cos(t1)*sin(phase));
    }
    else {
        return f0 / (sin(t0)*cos(phase) + cos(t0)*sin(phase));
    }
}

inline double fdtd::calc_phase(double t0, double t1, double t2, double f0, double f1, double f2)
{
    double f10 = f1 - f0,
           f21 = f2 - f1;

    if(f10 == 0 && f21 == 0) {
        return 0.0;
    }
    else {
        return atan2(f10*(sin(t2)-sin(t1)) - f21*(sin(t1)-sin(t0)),
                     f21*(cos(t1)-cos(t0)) - f10*(cos(t2)-cos(t1)));
    }
}

inline double fdtd::calc_amplitude(double t0, double t1, double t2, double f0, double f1, double f2, double phase)
{
    double f21 = f2 - f1,
           f10 = f1 - f0;

    if(f21 == 0 && f10 == 0) {
        return 0.0;
    }
    else if(f21*f21 >= f10*f10) {
        return f21 / (cos(phase)*(sin(t2)-sin(t1)) + sin(phase)*(cos(t2)-cos(t1)));
    }
    else {
        return f10 / (cos(phase)*(sin(t1)-sin(t0)) + sin(phase)*(cos(t1)-cos(t0)));
    }
}

///////////////////////////////////////////////////////////////////////////
// Source management
///////////////////////////////////////////////////////////////////////////
void fdtd::FDTD::add_source(complex128 *Jx, complex128 *Jy, complex128 *Jz,
                            complex128 *Mx, complex128 *My, complex128 *Mz,
                            int i0, int j0, int k0, int I, int J, int K,
                            bool calc_phase)
{

    // these source arrays may *actually* be compelx-valued. In the time
    // domain, complex values correspond to temporal phase shifts. We need
    // to convert the complex value to an amplitude and phase. Fortunately,
    // we can use the memory that is already allocated for these values.
    // Specifically, we use src_array.real = amplitude and
    // src_array.imag = phase
    //
    // Important note: EMopt assumes the time dependence is exp(-i*omega*t).
    // In order to account for this minus sign, we need to invert the sign
    // of the calculated phase.
    if(calc_phase) {
		int ind=0;
		double real, imag;

		for(int i = 0; i < I; i++) {
			for(int j = 0; j < J; j++) {
				for(int k = 0; k < K; k++) {
					ind = i*J*K + j*K + k;


					// Jx
					real = Jx[ind].real;
					imag = Jx[ind].imag;

					Jx[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) Jx[ind].imag = 0.0;
					else Jx[ind].imag = -1*atan2(imag, real);

					// Jy
					real = Jy[ind].real;
					imag = Jy[ind].imag;

					Jy[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) Jy[ind].imag = 0.0;
					else Jy[ind].imag = -1*atan2(imag, real);

					// Jz
					real = Jz[ind].real;
					imag = Jz[ind].imag;

					Jz[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) Jz[ind].imag = 0.0;
					else Jz[ind].imag = -1*atan2(imag, real);

					// Mx
					real = Mx[ind].real;
					imag = Mx[ind].imag;

					Mx[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) Mx[ind].imag = 0.0;
					else Mx[ind].imag = -1*atan2(imag, real);

					// My
					real = My[ind].real;
					imag = My[ind].imag;

					My[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) My[ind].imag = 0.0;
					else My[ind].imag = -1*atan2(imag, real);

					// Mz
					real = Mz[ind].real;
					imag = Mz[ind].imag;

					Mz[ind].real = sqrt(real*real + imag*imag);
					if(imag == 0 && real == 0) Mz[ind].imag = 0.0;
					else Mz[ind].imag = -1*atan2(imag, real);

				}
			}
		}
    }
	else {
		int N = I * J * K;
		complex128 *cuda_Jx, *cuda_Jy, *cuda_Jz, *cuda_Mx, *cuda_My, *cuda_Mz;
		hipMallocManaged((void **)&cuda_Jx, N*sizeof(complex128));
		hipMallocManaged((void **)&cuda_Jy, N*sizeof(complex128));
		hipMallocManaged((void **)&cuda_Jz, N*sizeof(complex128));
		hipMallocManaged((void **)&cuda_Mx, N*sizeof(complex128));
		hipMallocManaged((void **)&cuda_My, N*sizeof(complex128));
		hipMallocManaged((void **)&cuda_Mz, N*sizeof(complex128));
		memcpy(cuda_Jx, Jx, N*sizeof(complex128));
		memcpy(cuda_Jy, Jy, N*sizeof(complex128));
		memcpy(cuda_Jz, Jz, N*sizeof(complex128));
		memcpy(cuda_Mx, Mx, N*sizeof(complex128));
		memcpy(cuda_My, My, N*sizeof(complex128));
		memcpy(cuda_Mz, Mz, N*sizeof(complex128));

		SourceArray src = {cuda_Jx, cuda_Jy, cuda_Jz, cuda_Mx, cuda_My, cuda_Mz, i0, j0, k0, I, J, K};
		_sources.push_back(src);
	}
}

void fdtd::FDTD::clear_sources()
{
    for(auto const& src : _sources) {
		hipFree(src.Jx);
		hipFree(src.Jy);
		hipFree(src.Jz);
		hipFree(src.Mx);
		hipFree(src.My);
		hipFree(src.Mz);
	}
    _sources.clear();
}

void fdtd::FDTD::set_source_properties(double src_T, double src_min)
{
    _hcd.src_T = src_T;
    _hcd.src_min = src_min;
    _hcd.src_k = src_T*src_T / log((1+src_min)/src_min);
}

inline double fdtd::FDTD::src_func_t(double t, double phase)
{
    if(t <= _hcd.src_T)
        return sin(t + phase)*((1+_hcd.src_min) * exp(-(t-_hcd.src_T)*(t-_hcd.src_T) / _hcd.src_k) - _hcd.src_min);
    else
        return sin(t + phase);
}


///////////////////////////////////////////////////////////////////////////
// Boundary Conditions
///////////////////////////////////////////////////////////////////////////
void fdtd::FDTD::set_bc(char* newbc)
{
	_hcd.bc0 = newbc[0];
	_hcd.bc1 = newbc[1];
	_hcd.bc2 = newbc[2];
}

///////////////////////////////////////////////////////////////////////////
// ctypes interface
///////////////////////////////////////////////////////////////////////////

fdtd::FDTD* FDTD_new(int Nx, int Ny, int Nz)
{
    return new fdtd::FDTD(Nx, Ny, Nz);
}

void FDTD_set_wavelength(fdtd::FDTD* fdtd, double wavelength)
{
    fdtd->set_wavelength(wavelength);
}

void FDTD_set_physical_dims(fdtd::FDTD* fdtd,
                            double X, double Y, double Z,
                            double dx, double dy, double dz)
{
    fdtd->set_physical_dims(X, Y, Z, dx, dy, dz);
}

void FDTD_set_dt(fdtd::FDTD* fdtd, double dt)
{
    fdtd->set_dt(dt);
}

void FDTD_set_complex_eps(fdtd::FDTD* fdtd, bool complex_eps)
{
    fdtd->set_complex_eps(complex_eps);
}

void FDTD_update(fdtd::FDTD* fdtd, double start_time, int num_time_steps)
{
    fdtd->update(start_time, num_time_steps);
}

void FDTD_set_pml_widths(fdtd::FDTD* fdtd, int xmin, int xmax,
                                           int ymin, int ymax,
                                           int zmin, int zmax)
{
    fdtd->set_pml_widths(xmin, xmax, ymin, ymax, zmin, zmax);
}

void FDTD_set_pml_properties(fdtd::FDTD* fdtd, double sigma, double alpha,
                                               double kappa, double pow)
{
    fdtd->set_pml_properties(sigma, alpha, kappa, pow);
}

void FDTD_build_pml(fdtd::FDTD* fdtd)
{
    fdtd->build_pml();
}

void FDTD_reset_pml(fdtd::FDTD* fdtd)
{
    fdtd->reset_pml();
}

void FDTD_set_t0_arrays(fdtd::FDTD* fdtd,
                         complex128 *Ex_t0, complex128 *Ey_t0, complex128 *Ez_t0,
                         complex128 *Hx_t0, complex128 *Hy_t0, complex128 *Hz_t0)
{
    fdtd->set_t0_arrays(Ex_t0, Ey_t0, Ez_t0, Hx_t0, Hy_t0, Hz_t0);
}

void FDTD_set_t1_arrays(fdtd::FDTD* fdtd,
                         complex128 *Ex_t1, complex128 *Ey_t1, complex128 *Ez_t1,
                         complex128 *Hx_t1, complex128 *Hy_t1, complex128 *Hz_t1)
{
    fdtd->set_t1_arrays(Ex_t1, Ey_t1, Ez_t1, Hx_t1, Hy_t1, Hz_t1);
}

double FDTD_calc_phase_2T(double t0, double t1, double f0, double f1)
{
    return fdtd::calc_phase(t0, t1, f0, f1);
}

double FDTD_calc_amplitude_2T(double t0, double t1, double f0, double f1, double phase)
{
    return fdtd::calc_amplitude(t0, t1, f0, f1, phase);
}

double FDTD_calc_phase_3T(double t0, double t1, double t2, double f0, double f1, double f2)
{
    return fdtd::calc_phase(t0, t1, t2, f0, f1, f2);
}

double FDTD_calc_amplitude_3T(double t0, double t1, double t2, double f0, double f1, double f2, double phase)
{
    return fdtd::calc_amplitude(t0, t1, t2, f0, f1, f2, phase);
}

void FDTD_capture_t0_fields(fdtd::FDTD* fdtd)
{
    fdtd->capture_t0_fields();
}

void FDTD_capture_t1_fields(fdtd::FDTD* fdtd)
{
    fdtd->capture_t1_fields();
}


void FDTD_calc_complex_fields_2T(fdtd::FDTD* fdtd, double t0, double t1)
{
    fdtd->calc_complex_fields(t0, t1);
}

void FDTD_calc_complex_fields_3T(fdtd::FDTD* fdtd, double t0, double t1, double t2)
{
    fdtd->calc_complex_fields(t0, t1, t2);
}

void FDTD_add_source(fdtd::FDTD* fdtd,
                     complex128 *Jx, complex128 *Jy, complex128 *Jz,
                     complex128 *Mx, complex128 *My, complex128 *Mz,
                     int i0, int j0, int k0, int I, int J, int K, bool calc_phase)
{
    fdtd->add_source(Jx, Jy, Jz, Mx, My, Mz, i0, j0, k0, I, J, K, calc_phase);
}

void FDTD_clear_sources(fdtd::FDTD* fdtd)
{
    fdtd->clear_sources();
}

void FDTD_set_source_properties(fdtd::FDTD* fdtd, double src_T, double src_min)
{
    fdtd->set_source_properties(src_T, src_min);
}

double FDTD_src_func_t(fdtd::FDTD* fdtd, double t, double phase)
{
    return fdtd->src_func_t(t, phase);
}

void FDTD_set_bc(fdtd::FDTD* fdtd, char* newbc)
{
    fdtd->set_bc(newbc);
}
